// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision$
// $Date$
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

#include <stdio.h>
#include <stdlib.h>

#include "cuda_util.h"
#include "cudpp_globals.h"
#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_plan.h"

#include "kernel/listrank_kernel.cuh"

/**
 * @file
 * listrank_app.cu
 * 
 * @brief CUDPP application-level listrank routines
 */

/** \addtogroup cudpp_app 
 * @{
 */

/** @name ListRank Functions
 * @{
 */

/** @brief Perform Huffman encoding
 * 
 * @todo
 *
 */
void listRank(int                       *d_ranked_values,
              int                       *d_unranked_values,
              int                       *d_next_indices,
              size_t                    head,
              size_t                    numElements,
              const CUDPPListRankPlan   *plan)
{
    int step = 1;
    int cnt = 1;
    int* d_tmp = d_next_indices;

    // thread info -- kernel1
    int nThreads = LISTRANK_CTA_BLOCK;
    int tThreads = LISTRANK_TOTAL;
    int nBlocks  = tThreads/LISTRANK_CTA_BLOCK;

    dim3 grid_construct   (nBlocks,  1, 1);
    dim3 threads_construct(nThreads, 1, 1);

    // thread info -- kernel2
    tThreads = LISTRANK_MAX;
    nBlocks = tThreads/LISTRANK_CTA_BLOCK;
    dim3 grid_construct2   (nBlocks,  1, 1);
    dim3 threads_construct2(nThreads, 1, 1);


    while(step<LISTRANK_MAX)
    {
        // Each step doubles the number of threads added to pointer "chase"
        if(cnt%2 == 1)
        {
            // ping
            list_rank_kernel_soa_1<<< grid_construct, threads_construct >>>
                (d_ranked_values, d_unranked_values, d_tmp,
                plan->m_d_tmp1, plan->m_d_tmp2, step, head, numElements);
            d_tmp = plan->m_d_tmp3;
        }
        else
        {
            // pong
            list_rank_kernel_soa_1<<< grid_construct, threads_construct >>>
                (d_ranked_values, d_unranked_values, plan->m_d_tmp1,
                d_tmp, plan->m_d_tmp2, step, head, numElements);
        }
        step *= 2;
        cnt++;
    }

    // Out of threads to dispatch, each thread now keeps chasing pointer until
    // all lists are ranked
    if(LISTRANK_MAX < numElements)
    {
        if(cnt%2 == 0)
        {
            list_rank_kernel_soa_2<<< grid_construct2, threads_construct2 >>>
                (d_ranked_values, d_unranked_values, plan->m_d_tmp1, plan->m_d_tmp2, head, numElements);
            CUDA_SAFE_CALL(hipDeviceSynchronize());
        }
        else
        {
            list_rank_kernel_soa_2<<< grid_construct2, threads_construct2 >>>
                (d_ranked_values, d_unranked_values, d_tmp, plan->m_d_tmp2, head, numElements);
            CUDA_SAFE_CALL(hipDeviceSynchronize());
        }
    }
}

#ifdef __cplusplus
extern "C" 
{
#endif

/** @brief Allocate intermediate arrays used by ListRank.
 *
 * @todo
 *
 * @param [in,out] plan Pointer to CUDPPListRankPlan object containing
 *                      options and number of elements, which is used
 *                      to compute storage requirements, and within
 *                      which intermediate storage is allocated.
 */
void allocListRankStorage(CUDPPListRankPlan *plan)
{
    size_t numElts = plan->m_numElements;

    CUDA_SAFE_CALL(hipMalloc( (void**) &(plan->m_d_tmp1),     numElts*sizeof(int) ));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(plan->m_d_tmp2),     numElts*sizeof(int) ));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(plan->m_d_tmp3),     numElts*sizeof(int) ));
}

/** @brief Deallocate intermediate block arrays in a CUDPPListRankPlan object.
 *
 * @todo 
 *
 * @param[in,out] plan Pointer to CUDPPListRankPlan object initialized by allocListRankStorage().
 */
void freeListRankStorage(CUDPPListRankPlan *plan)
{
    if(plan->m_d_tmp1 != NULL) CUDA_SAFE_CALL(hipFree(plan->m_d_tmp1));
    if(plan->m_d_tmp2 != NULL) CUDA_SAFE_CALL(hipFree(plan->m_d_tmp2));
    if(plan->m_d_tmp3 != NULL) CUDA_SAFE_CALL(hipFree(plan->m_d_tmp3));
}


/** @brief Dispatch function to perform parallel list ranking on a
 * linked-list with the specified configuration.
 *
 * @todo
 * 
 * @param[out] d_ranked_values Ranked values array
 * @param[in]  d_unranked_values Unranked values array
 * @param[in]  d_next_indices Next indices array
 * @param[in]  head Head pointer index
 * @param[in]  numElements Number of nodes values to rank
 * @param[in]  plan     Pointer to CUDPPListRankPlan object containing
 *                      list ranking options and intermediate storage
 */
void cudppListRankDispatch(void *d_ranked_values,
                           void *d_unranked_values,
                           void *d_next_indices,
                           size_t head,
                           size_t numElements,
                           const CUDPPListRankPlan *plan)
{
    // Call to list ranker
    // TODO - template to allow other value types
    listRank((int*) d_ranked_values, (int*) d_unranked_values,
             (int*) d_next_indices, head, numElements, plan);
}


#ifdef __cplusplus
}
#endif


/** @} */ // end listrank functions
/** @} */ // end cudpp_app