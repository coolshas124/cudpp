// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision: 4400 $
// $Date: 2008-08-04 10:58:14 -0700 (Mon, 04 Aug 2008) $
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * tridiagonal_app.cu
 *
 * @brief CUDPP application-level tridiagonal solver routines
 */

/** \addtogroup cudpp_app
  * @{
  */
/** @name Tridiagonal functions
 * @{
 */

#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_plan.h"
#include "cuda_util.h"

#include <cstdlib>
#include <cstdio>
#include <assert.h>

#include "kernel/tridiagonal_kernel.cuh"

/**
 * @brief Hybrid CR-PCR solver (CRPCR)
 *
 * This is a wrapper function for the GPU CR-PCR kernel.
 *
 * @param[out] d_x Solution vector
 * @param[in] d_a Lower diagonal
 * @param[in] d_b Main diagonal
 * @param[in] d_c Upper diagonal
 * @param[in] d_d Right hand side
 * @param[in] systemSize The size of the linear system
 * @param[in] numSystems The number of systems to be solved
 */
template <class T>
void crpcr(T *d_a, 
           T *d_b, 
           T *d_c, 
           T *d_d, 
           T *d_x, 
           unsigned int systemSizeOriginal, 
           unsigned int numSystems)
{
    const unsigned int systemSize = ceilPow2(systemSizeOriginal);
    const unsigned int num_threads_block = systemSize/2;
    const unsigned int restSystemSize = systemSize/2;
    const unsigned int iterations = logBase2Pow2(restSystemSize/2);
  
    // setup execution parameters
    dim3  grid(numSystems, 1, 1);
    dim3  threads(num_threads_block, 1, 1);
    const unsigned int smemSize = 
        (systemSize + 1 + restSystemSize) * 5 * sizeof(T);

    crpcrKernel<<< grid, threads, smemSize>>>(d_a, 
                                              d_b, 
                                              d_c, 
                                              d_d, 
                                              d_x, 
                                              systemSizeOriginal,
                                              iterations);

    CUDA_CHECK_ERROR("crpcr");
}


/**
 * @brief Dispatches the tridiagonal function based on the plan
 *
 * This is the dispatch call for the tridiagonal solver in either float 
 * or double datatype. 
 *
 * @param[out] d_x Solution vector
 * @param[in] d_a Lower diagonal
 * @param[in] d_b Main diagonal
 * @param[in] d_c Upper diagonal
 * @param[in] d_d Right hand side
 * @param[in] systemSize The size of the linear system
 * @param[in] numSystems The number of systems to be solved
 * @param[in] plan pointer to CUDPPTridiagonalPlan
 * @returns CUDPPResult indicating success or error condition
 */
CUDPPResult cudppTridiagonalDispatch(void *d_a, 
                                     void *d_b, 
                                     void *d_c, 
                                     void *d_d, 
                                     void *d_x, 
                                     int systemSize, 
                                     int numSystems, 
                                     const CUDPPTridiagonalPlan * plan)
{
  
    //figure out which algorithm to run
    if (plan->m_config.datatype == CUDPP_FLOAT)
    {
        crpcr<float>((float *)d_a, 
                     (float *)d_b, 
                     (float *)d_c, 
                     (float *)d_d, 
                     (float *)d_x, 
                     systemSize, 
                     numSystems);
        return CUDPP_SUCCESS;
    }
    else if (plan->m_config.datatype == CUDPP_DOUBLE)
    {
        crpcr<double>((double *)d_a, 
                      (double *)d_b, 
                      (double *)d_c, 
                      (double *)d_d, 
                      (double *)d_x, 
                      systemSize, 
                      numSystems);
        return CUDPP_SUCCESS;
    }
    else
        return CUDPP_ERROR_ILLEGAL_CONFIGURATION;
    
}

/** @} */ // end Tridiagonal functions
/** @} */ // end cudpp_app
