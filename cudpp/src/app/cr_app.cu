// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
//  $Revision:
//  $Date:
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * cr_app.cu
 *
 * @brief CUDPP application-level CR tridiagonal solver
 */

/** \addtogroup cudpp_app
  * @{
  */
/** @name Cyclic reduction solver (CR)
 * @{
 */

#include "kernel/cr_kernel.cu"

/**
 * @brief Cyclic reduction solver (CR)
 *
 * This is a wrapper function for the GPU CR kernel.
 *
 * @param[out] d_x Solution vector
 * @param[in] d_a Lower diagonal
 * @param[in] d_b Main diagonal
 * @param[in] d_c Upper diagonal
 * @param[in] d_d Right hand side
 * @param[in] systemSize The size of the linear system
 * @param[in] numSystems The number of systems to be solved
 */

template <class T>
void cr(T *a, T *b, T *c, T *d, T *x, int systemSize, int numSystems)
{
    const unsigned int num_threads_block = systemSize/2;
    const unsigned int memSize = sizeof(T)*numSystems*systemSize;

    // allocate device memory input and output arrays
    T* d_a;
    T* d_b;
    T* d_c;
    T* d_d;
    T* d_x;

    unsigned int timer;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    cutStartTimer(timer);

    CUDA_SAFE_CALL( hipMalloc( (void**) &d_a,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_b,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_c,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_d,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_x,memSize));

    cutStopTimer(timer);
    printf("GPU hipMalloc time: %f ms\n", cutGetTimerValue(timer));

   // copy host memory to device input array
    CUDA_SAFE_CALL( hipMemcpy( d_a, a,memSize, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( d_b, b,memSize, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( d_c, c,memSize, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( d_d, d,memSize, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( d_x, x,memSize, hipMemcpyHostToDevice));

    // setup execution parameters
    dim3  grid(numSystems, 1, 1);
    dim3  threads(num_threads_block, 1, 1);

    crKernel<<< grid, threads,systemSize*5*sizeof(T)>>>(d_a, d_b, d_c, d_d, d_x);
    //hipDeviceSynchronize();

    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy(x, d_x,memSize, hipMemcpyDeviceToHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_b));
    CUDA_SAFE_CALL(hipFree(d_c));
    CUDA_SAFE_CALL(hipFree(d_d));
    CUDA_SAFE_CALL(hipFree(d_x));
}
/** @} */ // end Cyclic reduction solver (CR)
/** @} */ // end cudpp_app

