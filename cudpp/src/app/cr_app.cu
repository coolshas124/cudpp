// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
//  $Revision:
//  $Date:
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * cr_app.cu
 *
 * @brief CUDPP application-level CR tridiagonal solver
 */

/** \addtogroup cudpp_app
  * @{
  */
/** @name Cyclic reduction solver (CR)
 * @{
 */

// #include "stopwatch.h"
#include "kernel/cr_kernel.cu"

/**
 * @brief Cyclic reduction solver (CR)
 *
 * This is a wrapper function for the GPU CR kernel.
 *
 * @param[out] d_x Solution vector
 * @param[in] d_a Lower diagonal
 * @param[in] d_b Main diagonal
 * @param[in] d_c Upper diagonal
 * @param[in] d_d Right hand side
 * @param[in] systemSize The size of the linear system
 * @param[in] numSystems The number of systems to be solved
 */

template <class T>
void cr(T *d_a, T *d_b, T *d_c, T *d_d, T *d_x, int systemSize, int numSystems)
{
    const unsigned int num_threads_block = systemSize/2;

    // setup execution parameters
    dim3  grid(numSystems, 1, 1);
    dim3  threads(num_threads_block, 1, 1);

    crKernel<<< grid, threads,systemSize*5*sizeof(T)>>>(d_a, d_b, d_c, d_d, d_x);
}
/** @} */ // end Cyclic reduction solver (CR)
/** @} */ // end cudpp_app

