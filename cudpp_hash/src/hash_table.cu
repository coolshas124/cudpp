#include "hip/hip_runtime.h"
#include "hash_table.h"
#include "debugging.cuh"

#include <mt19937ar.h>

#include <cstdio>
#include <cstring>
#include <limits>

#include <hip/hip_runtime.h>
#include "cuda_util.h"

namespace CudaHT {
namespace CuckooHashing {

char buffer[256];

//! @name Internal
/// @{
dim3 ComputeGridDim(unsigned n) {
    // Round up in order to make sure all items are hashed in.
    dim3 grid( (n + kBlockSize-1) / kBlockSize );
    if (grid.x > kGridSize) {
        grid.y = (grid.x + kGridSize - 1) / kGridSize;
        grid.x = kGridSize;
    }
    return grid;
}


unsigned ComputeMaxIterations(const unsigned n,
                              const unsigned table_size,
                              const unsigned num_functions) {
    float lg_input_size = log(n) / log(2.0f);

// #define CONSTANT_ITERATIONS
#ifdef CONSTANT_ITERATIONS
    // Set the maximum number of iterations to 7lg(N).
    const unsigned MAX_ITERATION_CONSTANT = 7;
    unsigned max_iterations = MAX_ITERATION_CONSTANT * lg_input_size;
#else
    // Use an empirical formula for determining what the maximum number of
    // iterations should be.  Works OK in most situations.
    float load_factor = float(n) / table_size;
    float ln_load_factor = log(load_factor) / log(2.71828183);

    unsigned max_iterations = 4 * ceil(-1.0 / (0.028255 + 1.1594772 * 
                                               ln_load_factor)* lg_input_size);
#endif

    // Debugging: Print out how many iterations are allowed.
#ifdef _DEBUG  
    sprintf(buffer, "Maximum iterations: %u (%f lg (N))", max_iterations, 
            max_iterations / lg_input_size);
    PrintMessage(buffer, false);

    sprintf(buffer, "Number of functions: %u", num_functions);
    PrintMessage(buffer, false);
#endif

    return max_iterations;
}
/// @}


bool HashTable::Initialize(const unsigned max_table_entries,
                           const float    space_usage,
                           const unsigned num_functions) {
    Release();

    // Determine the minimum amount of slots the table requires,
    // and whether the space_usage is within range.
    float minimum_space_usage;
    if (num_functions < 2 || num_functions > 5) {
        char message[256] = "Number of hash functions must be from 2 to 5; "
            "others are unimplemented.";
        PrintMessage(message, true);
        return false;
    } else {
        minimum_space_usage = kMinimumSpaceUsages[num_functions];
    }

    if (space_usage < minimum_space_usage) {
        sprintf(buffer, "Minimum possible space usage for %u functions is %f.",
                num_functions, minimum_space_usage);
        PrintMessage(buffer);
        return false;
    }

    num_hash_functions_ = num_functions;
    table_size_ = unsigned(ceil(max_table_entries * space_usage));

#ifdef _DEBUG
    sprintf(buffer, "Table size: %u slots", table_size_);
    PrintMessage(buffer);
#endif

    // Allocate memory.
    const unsigned slots_to_allocate = table_size_ + kStashSize;
    CUDA_SAFE_CALL(hipMalloc( (void**)&d_contents_, 
                               sizeof(Entry) * slots_to_allocate ));
    CUDA_SAFE_CALL(hipMalloc( (void**)&d_failures_, sizeof(unsigned) ));
    if (!d_contents_ || !d_failures_) {
        fprintf(stderr, "Failed to allocate %u slots.\n", slots_to_allocate);
        return false;
    }

    return true;
}


void HashTable::Release() {
    table_size_  = 0;

    hipFree(d_contents_);
    hipFree(d_failures_);

    d_contents_ = NULL;
    d_failures_ = NULL;
}


bool HashTable::Build(const unsigned  n,
                      const unsigned *d_keys,
                      const unsigned *d_values) {
    unsigned max_iterations = ComputeMaxIterations(n, table_size_, 
                                                   num_hash_functions_);
    unsigned num_failures = 1;
    unsigned num_attempts = 0;

    // Storage for statistics collection.
    unsigned *d_iterations_taken = NULL;
#ifdef TRACK_ITERATIONS
    CUDA_SAFE_CALL(hipMalloc((void**)&d_iterations_taken, sizeof(unsigned) * n));
#endif

    // Track how many items ended up in the stash.
    unsigned *d_stash_count = NULL;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_stash_count, sizeof(unsigned)));

    // Main build loop.
    while (num_failures && ++num_attempts < kMaxRestartAttempts) {
        CUDA_SAFE_CALL(hipMemset(d_stash_count, 0, sizeof(unsigned)));
    
        // Generate new hash functions.
        if (num_hash_functions_ == 2)
            constants_2_.Generate(n, d_keys,table_size_);
        else if (num_hash_functions_ == 3)
            constants_3_.Generate(n, d_keys,table_size_);
        else if (num_hash_functions_ == 4)
            constants_4_.Generate(n, d_keys,table_size_);
        else
            constants_5_.Generate(n, d_keys,table_size_);

        stash_constants_.x = std::max(1lu, genrand_int32()) % kPrimeDivisor;
        stash_constants_.y = genrand_int32() % kPrimeDivisor;
        stash_count_ = 0;

        // Initialize memory.
        unsigned slots_in_table = table_size_ + kStashSize;
        clear_table<<<ComputeGridDim(slots_in_table), kBlockSize>>>
            (slots_in_table, kEntryEmpty, d_contents_);
        CUDA_CHECK_ERROR("Error occurred during hash table clear.\n");
        num_failures = 0;

        // Build the table.
        CUDA_SAFE_CALL(hipMemset(d_failures_, 0, sizeof(unsigned)));
        if (num_hash_functions_ == 2) {
            CuckooHash<<<ComputeGridDim(n), kBlockSize>>>
                (n,
                 d_keys,
                 d_values,
                 table_size_,
                 constants_2_,
                 max_iterations,
                 d_contents_,
                 stash_constants_,
                 d_stash_count,
                 d_failures_,
                 d_iterations_taken);
        } else if (num_hash_functions_ == 3) {
            CuckooHash<<<ComputeGridDim(n), kBlockSize>>>
                (n,
                 d_keys,
                 d_values,
                 table_size_,
                 constants_3_,
                 max_iterations,
                 d_contents_,
                 stash_constants_,
                 d_stash_count,
                 d_failures_,
                 d_iterations_taken);
        } else if (num_hash_functions_ == 4) {
            CuckooHash<<<ComputeGridDim(n), kBlockSize>>>
                (n,
                 d_keys,
                 d_values,
                 table_size_,
                 constants_4_,
                 max_iterations,
                 d_contents_,
                 stash_constants_,
                 d_stash_count,
                 d_failures_,
                 d_iterations_taken);
        } else {
            CuckooHash<<<ComputeGridDim(n), kBlockSize>>>
                (n,
                 d_keys,
                 d_values,
                 table_size_,
                 constants_5_,
                 max_iterations,
                 d_contents_,
                 stash_constants_,
                 d_stash_count,
                 d_failures_,
                 d_iterations_taken);
        }
    
        CUDA_CHECK_ERROR("Error occurred during hash table build.\n");

        // Check if successful.
        hipMemcpy( &num_failures, d_failures_, sizeof(unsigned), hipMemcpyDeviceToHost );

#ifdef COUNT_UNINSERTED
        if (num_failures) {
            printf("Failed to insert %u items.\n", num_failures);
        }
#endif
    }

    // Copy out the stash size.
    hipMemcpy( &stash_count_, d_stash_count, sizeof(unsigned), hipMemcpyDeviceToHost );
    if (stash_count_ && num_failures == 0) {
        sprintf(buffer, "Stash size: %u", stash_count_);
        PrintMessage(buffer, true);

#ifdef _DEBUG
        PrintStashContents(d_contents_ + table_size_);
#endif    
    }
    CUDA_SAFE_CALL(hipFree(d_stash_count));

#ifdef TRACK_ITERATIONS
    if (num_failures == 0) {
        OutputBuildStatistics(n, d_iterations_taken);
    }
    CUDA_SAFE_CALL(hipFree(d_iterations_taken));
#endif

    // Dump some info if a restart was required.
    if (num_attempts >= kMaxRestartAttempts) {
        sprintf(buffer, "Completely failed to build");
        PrintMessage(buffer, true);
    } else if (num_attempts > 1) {
        sprintf(buffer, "Needed %u attempts to build", num_attempts);
        PrintMessage(buffer, true);
    }

    CUDA_CHECK_ERROR("Error occurred during hash table build.\n");
    return num_failures == 0;
}


void HashTable::Retrieve(const unsigned  n_queries,
                         const unsigned *d_keys,
                         unsigned *d_values) {
    unsigned *d_retrieval_probes = NULL;
#ifdef TRACK_ITERATIONS
    CUDA_SAFE_CALL(hipMalloc((void**)&d_retrieval_probes, sizeof(unsigned) * n_queries));
#endif

    if (num_hash_functions_ == 2) {
        hash_retrieve<<<ComputeGridDim(n_queries), kBlockSize>>>
            (n_queries,
             d_keys,
             table_size_,
             d_contents_,
             constants_2_,
             stash_constants_,
             stash_count_,
             d_values,
             d_retrieval_probes);
    } else if (num_hash_functions_ == 3) {
        hash_retrieve<<<ComputeGridDim(n_queries), kBlockSize>>>
            (n_queries,
             d_keys,
             table_size_,
             d_contents_,
             constants_3_,
             stash_constants_,
             stash_count_,
             d_values,
             d_retrieval_probes);
    } else if (num_hash_functions_ == 4) {
        hash_retrieve<<<ComputeGridDim(n_queries), kBlockSize>>>
            (n_queries,
             d_keys,
             table_size_,
             d_contents_,
             constants_4_,
             stash_constants_,
             stash_count_,
             d_values,
             d_retrieval_probes);
    } else {
        hash_retrieve<<<ComputeGridDim(n_queries), kBlockSize>>>
            (n_queries,
             d_keys,
             table_size_,
             d_contents_,
             constants_5_,
             stash_constants_,
             stash_count_,
             d_values,
             d_retrieval_probes);
    }
  
    CUDA_CHECK_ERROR("Retrieval failed.\n");

#ifdef TRACK_ITERATIONS
    OutputRetrievalStatistics(n_queries,
                              d_retrieval_probes,
                              num_hash_functions_);
    CUDA_SAFE_CALL(hipFree(d_retrieval_probes));
#endif
}


};  // namesapce CuckooHashing
};  // namespace CudaHT


// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
